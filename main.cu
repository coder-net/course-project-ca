#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <tuple>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "hipblas.h"
#pragma comment(lib, "cublas.lib")

#include "cuda_timer.cuh"
#include "utils.cuh"
#include "algorithm1.cuh"
#include "algorithm2.cuh"
#include "strassen.cuh"



std::tuple<float*, size_t, size_t> matrixMultiplicationWithCuda(float* a, size_t a_row, size_t a_col, float* b, size_t b_row, size_t b_col) {
	float* cuda_a = nullptr;
	float* cuda_b = nullptr;
	float* cuda_c = nullptr;
	size_t a_size = a_row * a_col;
	size_t b_size = b_row * b_col;
	size_t c_size = a_row * b_col;

	hipMalloc((void**)&cuda_a, sizeof(float) * a_size);
	hipMalloc((void**)&cuda_b, sizeof(float) * b_size);
	hipMalloc((void**)&cuda_c, sizeof(float) * c_size);

	hipMemcpy(cuda_a, a, sizeof(float) * a_size, hipMemcpyHostToDevice);
	hipMemcpy(cuda_b, b, sizeof(float) * b_size, hipMemcpyHostToDevice);

	cudaMatrixMultiplication(cuda_a, a_row, a_col, cuda_b, b_row, b_col, cuda_c, 1);

	float* temp_c = (float*)malloc(sizeof(float) * c_size);
	float* c = (float*)malloc(sizeof(float*) * c_size);

	hipMemcpy(temp_c, cuda_c, sizeof(float) * c_size, hipMemcpyDeviceToHost);

	toRowMajor(temp_c, c, a_row, b_col);

	free(temp_c);

	hipFree(cuda_a);
	hipFree(cuda_b);
	hipFree(cuda_c);

  return std::tie(c, a_row, b_col);
}


int main()
{
  float* A;
  float* B;
  float* C;
  size_t a_rows, a_cols, b_rows, b_cols, c_rows, c_cols;

  std::tie(A, a_rows, a_cols) = readMatrixFromFile("matrix1.txt");
  std::tie(B, b_rows, b_cols) = readMatrixFromFile("matrix2.txt");

  if (a_cols != b_rows) {
    std::cout << "Impossible to multiply these two matrix";
    return -1;
  }

  std::cout << "A: " << std::endl;
  printMatrix(A, a_rows, a_cols);
  std::cout << std::endl << "B: " << std::endl;
  printMatrix(B, b_rows, b_cols);
  std::cout << std::endl;

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);

  float start = clock();
  std::tie(C, c_rows, c_cols) = matrixMultiplication(A, a_rows, a_cols, B, b_rows, b_cols);
  float cpu_time = (clock() - start) / CLOCKS_PER_SEC;

  std::cout << "CPU time: " << cpu_time << "sec" << std::endl;

	if (cudaStatus != hipSuccess) {
	  fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
  }
  else {
    CudaTimer timer;

    timer.start();
    std::tie(C, c_rows, c_cols) = matrixMultiplicationWithCuda(A, a_rows, a_cols, B, b_rows, b_cols);
    timer.stop();

    float time = timer.value();
    
    timer.start();
    std::tie(C, c_rows, c_cols) = partialMatrixMultiplication1(A, a_rows, a_cols, B, b_rows, b_cols);
    timer.stop();

    float algo1_time = timer.value() ;

    writeMatrixToFile("algorithm1_out.txt", C, c_rows, c_cols);

    free(C);
    
    timer.start();
    std::tie(C, c_rows, c_cols) = partialMatrixMultiplication2(A, a_rows, a_cols, B, b_rows, b_cols);
    timer.stop();
    
    float algo2_time = timer.value();
 
    std::cout << "Simple multiplication: " << time << "sec" << std::endl
              << "Algo1 time: " << algo1_time << "sec" << std::endl 
              << "Algo2 time: " << algo2_time << "sec" << std::endl;

  }

  printMatrix(C, c_rows, c_cols);

  free(A);
  free(B);
  free(C);

	return 0;
}