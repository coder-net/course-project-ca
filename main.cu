#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <tuple>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "hipblas.h"
#pragma comment(lib, "cublas.lib")

#include "cuda_timer.cuh"
#include "utils.cuh"
#include "algorithm1.cuh"
#include "algorithm2.cuh"
#include "matrix_multiplication.cuh"

#include "strassen.cuh"



std::tuple<float*, size_t, size_t> matrixMultiplicationWithCuda(float* a, size_t a_row, size_t a_col, float* b, size_t b_row, size_t b_col) {
	float* cuda_a = nullptr;
	float* cuda_b = nullptr;
	float* cuda_c = nullptr;
	size_t a_size = a_row * a_col;
	size_t b_size = b_row * b_col;
	size_t c_size = a_row * b_col;

	hipMalloc((void**)&cuda_a, sizeof(float) * a_size);
	hipMalloc((void**)&cuda_b, sizeof(float) * b_size);
	hipMalloc((void**)&cuda_c, sizeof(float) * c_size);

	hipMemcpy(cuda_a, a, sizeof(float) * a_size, hipMemcpyHostToDevice);
	hipMemcpy(cuda_b, b, sizeof(float) * b_size, hipMemcpyHostToDevice);

	cudaMatrixMultiplication(cuda_a, a_row, a_col, cuda_b, b_row, b_col, cuda_c, 1);

	float* temp_c = (float*)malloc(sizeof(float) * c_size);
	float* c = (float*)malloc(sizeof(float*) * c_size);

	hipMemcpy(temp_c, cuda_c, sizeof(float) * c_size, hipMemcpyDeviceToHost);

	toRowMajor(temp_c, c, a_row, b_col);

	free(temp_c);

	hipFree(cuda_a);
	hipFree(cuda_b);
	hipFree(cuda_c);

  return std::tie(c, a_row, b_col);
}


int main()
{
  double* h_A;
  double* h_B;
  double* h_C;
  size_t a_rows, a_cols, b_rows, b_cols, c_rows, c_cols;
  
  std::tie(h_A, a_rows, a_cols) = readMatrixFromFile("matrix3.txt");
  std::tie(h_B, b_rows, b_cols) = readMatrixFromFile("matrix4.txt");

  size_t M = a_rows, K = a_cols, N = b_cols;

  h_C = (double*)malloc(sizeof(double) * M * N);

  strassen_mm(h_A, h_B, h_C, M, K,N);

  printMatrix(h_C, M, N);
  //int iter = 1;
  //int check = 0;
  //int depth = 2;

  //int sizeA = M * K;
  //int sizeB = K * N;
  //int sizeC = M * N;
  //int memSizeA = sizeA * sizeof(double);
  //int memSizeB = sizeB * sizeof(double);
  //int memSizeC = sizeC * sizeof(double);

  //double *h_C = (double *)malloc(memSizeC);

  //printMatrix(h_A, M, K);
  //printMatrix(h_B, K, N);

  //double *d_A, *d_B, *d_C;
  //hipMalloc((void**)&d_A, memSizeA);
  //hipMalloc((void**)&d_B, memSizeB);
  //hipMalloc((void**)&d_C, memSizeC);
  //hipMemcpy(d_A, h_A, memSizeA, hipMemcpyHostToDevice);
  //hipMemcpy(d_B, h_B, memSizeB, hipMemcpyHostToDevice);



  //CudaTimer ct;
  //ct.start();
  //  strassen(d_A, d_B, d_C, K, N, N, K, N, N, M, K, M, 2);
  //ct.stop();

  //double strassenTime = ct.value() / iter;
  //hipMemcpy(h_C, d_C, memSizeC, hipMemcpyDeviceToHost);

  //printMatrix(h_C, M, N);
//  if (a_cols != b_rows) {
//    std::cout << "Impossible to multiply these two matrix";
//    return -1;
//  }
//
//  std::cout << "A: " << std::endl;
//  printMatrix(A, a_rows, a_cols);
//  std::cout << std::endl << "B: " << std::endl;
//  printMatrix(B, b_rows, b_cols);
//  std::cout << std::endl;
//
//	hipError_t cudaStatus;
//	cudaStatus = hipSetDevice(0);
//
//  float start = clock();
//  //std::tie(C, c_rows, c_cols) = matrixMultiplication(A, a_rows, a_cols, B, b_rows, b_cols);
//  float cpu_time = (clock() - start) / CLOCKS_PER_SEC;
//
//  std::cout << "CPU time: " << cpu_time << "sec" << std::endl;
//
//	if (cudaStatus != hipSuccess) {
//	  fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//  }
//  else {
////    done_matrix_square(A, B, C, a_rows, a_cols, b_cols, 2);
//    CudaTimer timer;
//
//
//
//
//    double* A_d;
//    double* B_d;
//    double* C_d;
//
//hipMalloc((void**)&A_d, sizeof(double) * a_rows * a_cols);
//hipMalloc((void**)&B_d, sizeof(double) * b_rows * b_cols);
//hipMalloc((void**)&C_d, sizeof(double) * N * M);
//hipMemcpy(A_d, A, sizeof(double) * M * K, hipMemcpyHostToDevice);
//hipMemcpy(B_d, B, sizeof(double) * N * K, hipMemcpyHostToDevice);
//// hipMemcpy(C_d, C, sizeof(float) * M * N, hipMemcpyHostToDevice);
//
//    timer.start();
//    strassen(A_d,  B_d, C_d, K, N, N, K, N, N, M,K, M, 5);
//hipDeviceSynchronize();
//    timer.stop();
//
//    float time = timer.value();
//    
//    /*timer.start();
//    std::tie(C, c_rows, c_cols) = matrixMultiplicationInParts(A, a_rows, a_cols, B, b_rows, b_cols, 5, 5);
//    timer.stop();
//
//    float algo1_time = timer.value() ;*/
//
//    C = (double*)malloc(sizeof(double) * N * N);
//    hipMemcpy(C, C_d, sizeof(double) * M * N, hipMemcpyDeviceToHost);
//
//  }
//
//  printMatrix(C, M, N);
//
//  free(A);
//  free(B);
//  // free(C);

	return 0;
}