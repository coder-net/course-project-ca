#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <tuple>
#include <stdio.h>
#include <stdlib.h>
#include "hipblas.h"
#pragma comment(lib, "cublas.lib")

#include "utils.cuh"
#include "cpp_utils.h"




std::tuple<float*, size_t, size_t> matrixMultiplicationWithCuda(float* a, size_t a_row, size_t a_col, float* b, size_t b_row, size_t b_col) {
	float* cuda_a = nullptr;
	float* cuda_b = nullptr;
	float* cuda_c = nullptr;
	size_t a_size = a_row * a_col;
	size_t b_size = b_row * b_col;
	size_t c_size = a_row * b_col;

	hipMalloc((void**)&cuda_a, sizeof(float) * a_size);
	hipMalloc((void**)&cuda_b, sizeof(float) * b_size);
	hipMalloc((void**)&cuda_c, sizeof(float) * c_size);

	// print(temp_a, a_size);

	hipMemcpy(cuda_a, a, sizeof(float) * a_size, hipMemcpyHostToDevice);
	hipMemcpy(cuda_b, b, sizeof(float) * b_size, hipMemcpyHostToDevice);

	cudaMatrixMultiplication(cuda_a, a_row, a_col, cuda_b, b_row, b_col, cuda_c, 1);

	float* temp_c = (float*)malloc(sizeof(float) * c_size);
	float* c = (float*)malloc(sizeof(float*) * c_size);

	hipMemcpy(temp_c, cuda_c, sizeof(float) * c_size, hipMemcpyDeviceToHost);

	toRowMajor(temp_c, c, a_row, b_col);

	free(temp_c);

	hipFree(cuda_a);
	hipFree(cuda_b);
	hipFree(cuda_c);

  return std::tie(c, a_row, b_col);
}



int main()
{
	hipError_t cudaStatus;
	
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
	    fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return 1;
	}

  float* A;
  float* B;
  float* C;
  size_t a_rows, a_cols, b_rows, b_cols, c_rows, c_cols;

  std::tie(A, a_rows, a_cols) = readMatrixFromFile("matrix1.txt.txt");
  std::tie(B, b_rows, b_cols) = readMatrixFromFile("matrix2.txt.txt");

	std::tie(C, c_rows, c_cols) = matrixMultiplicationWithCuda(A, a_rows, a_cols, B, b_cols, b_rows);

	cudaStatus = hipDeviceSynchronize();
	
  printMatrix(C, c_rows, c_cols);

	return 0;
}